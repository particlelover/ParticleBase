#include "cudaParticleRotation.hh"
#include "kernelfuncs.h"
#include "kerneltemplate.hh"

#include <iostream>

cudaParticleRotation::~cudaParticleRotation()
{
  if (w != NULL)
    hipFree(w);
  if (m0inv != NULL)
    hipFree(m0inv);
  if (L != NULL)
    hipFree(L);
  if (T != NULL)
    hipFree(T);
}

void cudaParticleRotation::setup(int n)
{

  hipMalloc((void **)&w, sizeof(real) * 3 * N);
  if (withInfo)
    ErrorInfo("malloc w[] on GPU");

  hipMalloc((void **)&m0inv, sizeof(real) * N);
  if (withInfo)
    ErrorInfo("malloc m0inv[] on GPU");

  hipMalloc((void **)&L, sizeof(real) * 3 * N);
  if (withInfo)
    ErrorInfo("malloc L[] on GPU");

  hipMalloc((void **)&T, sizeof(real) * 3 * N);
  if (withInfo)
    ErrorInfo("malloc T[] on GPU");
}

void cudaParticleRotation::TimeEvolution(real dt)
{
  propagateEulerianRotation<<<MPnum, THnum1D>>>(w, dt, L, T, m0inv, move, N);

  if (withInfo)
    ErrorInfo("do TimeEvolution Eulerian Equation of Motion");
}

void cudaParticleRotation::setInertia(real r0_all)
{
  std::vector<real> r0(N, r0_all);
  setInertia(r0);
}

void cudaParticleRotation::setInertia(const std::vector<real> &_r0)
{
  clearArray<<<MPnum, THnum1D>>>(w, N * 3);
  clearArray<<<MPnum, THnum1D>>>(L, N * 3);

  hipMemcpy(m0inv, minv, sizeof(real) * N, hipMemcpyDeviceToDevice);

  std::vector<real> r1(N);
  for (int i = 0; i < N; ++i)
    r1[i] = 1 / (0.4 * _r0[i] * _r0[i]);

  hipMemcpy(tmp3N, &(r1[0]), sizeof(real) * N, hipMemcpyHostToDevice);
  multiplies<<<MPnum, THnum1D>>>(m0inv, tmp3N, N);

  if (withInfo)
    ErrorInfo("set Initial Inertia");
}
