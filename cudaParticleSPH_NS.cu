#include "hip/hip_runtime.h"
#include "cudaParticleSPH_NS.hh"
#include "kernelfuncs.h"
#include "kerneltemplate.hh"
#include <assert.h>
#include <boost/lexical_cast.hpp>

cudaParticleSPH_NS::~cudaParticleSPH_NS() {
  if (mu0!=NULL) hipFree(mu0);
  if (c2!=NULL)  hipFree(c2);
  //if (mu!=NULL)  hipFree(mu);
}

void cudaParticleSPH_NS::setup(int n) {
  offset_additional_output = n;

  cudaParticleSPHBase::setup(n);

  // for output x, y, z, num, rho, mu
  TMP.resize(n*8);

  // alloc mu0
  hipMalloc((void **)&mu0, sizeof(real)*N);
  hipMalloc((void **)&c2, sizeof(real)*N);
  if (withInfo) ErrorInfo("malloc mu0[], c2[], mu[] on GPU");

  clearArray<<<4, 256>>>(mu0, N, (real)1.0);
}

void cudaParticleSPH_NS::getPosition(void) {
  size_t sizeN = sizeof(float4) * N;

  pthread_mutex_lock(&mutTMP);
  hipMemcpy(&(TMP[0]),   r, sizeN, hipMemcpyDeviceToHost);   // r.x, r.y, r.z for position
  hipMemcpy(&(TMP[N*4]), num, sizeN, hipMemcpyDeviceToHost); // num.x, num.y, num.z for num, rho, mu
  pthread_mutex_unlock(&mutTMP);

  if (withInfo) ErrorInfo("cudaParticleSPH_NS::getPosition");
}

void cudaParticleSPH_NS::getSelectedPosition(void) {
  offset_additional_output = selected[1];
  const size_t sizeN = sizeof(float4) * selected[1];

  pthread_mutex_lock(&mutTMP);
  hipMemcpy(&(TMP[0]), &(r[selected[0]]), sizeN, hipMemcpyDeviceToHost);  // r.x, r.y, r.z for position
  hipMemcpy(&(TMP[selected[1]*4]), &(num[selected[0]]), sizeN, hipMemcpyDeviceToHost);  // num.x, num.y, num.z for num, rho, mu
  pthread_mutex_unlock(&mutTMP);

  if (withInfo) ErrorInfo("do getPosition (selected)");
}

void cudaParticleSPH_NS::setSPHProperties(const std::valarray<real> &_mu0, std::valarray<real> _c1, real _h){
  cudaParticleSPHBase::setSPHProperties(_h);

  const size_t sizeN = sizeof(real) * N;

  hipMemcpy(mu0, &(_mu0[0]), sizeN, hipMemcpyHostToDevice);

  // copy (sound velocity)^2
  _c1 *= _c1;
  hipMemcpy(c2, &(_c1[0]), sizeN, hipMemcpyHostToDevice);
}

std::string cudaParticleSPH_NS::additionalOutput(uint32_t i) const {
  std::string r = " " + boost::lexical_cast<std::string>(TMP[i*4 + offset_additional_output*4]) +
    " " + boost::lexical_cast<std::string>(TMP[i*4+1 + offset_additional_output*4]) +
    " " + boost::lexical_cast<std::string>(TMP[i*4+2 + offset_additional_output*4]);
  return r;
}

void cudaParticleSPH_NS::calcAcceleration(bool sortedOutput) {
  //real n_threshold = inspectDensity()*0.96;

  // at first clear a[]
  clearArray_F4<<<MPnum, THnum1D>>>(a, N);

#if defined(CUDACUTOFFBLOCK)
  dim3 _mpnum, _thnum;
  //_thnum.x = THnum2D; _thnum.y = THnum2D; _thnum.z = 1;
  _thnum.x = 8; _thnum.y = 9; _thnum.z = 1;
  assert(_thnum.x*_thnum.y <= threadsMax);
  _mpnum.x = myBlockNum; _mpnum.y = 3; _mpnum.z = 1;
  clearArray<<<MPnum, THnum1D>>>(tmp81N, N*81);
  class SPHNavierStokes<SPHKernelLucyDW> P;
  P.v = v;
  P.m = m;
  P.rhoinv = rhoinv;
  P.num = num;
  P.c2  = c2;
  P.h = h; P.w1 =-12*w0;
  P.rho0 = 1.0;

  int validOffset = 0;
  uint32_t *Q = NULL;
  uint32_t blocks2calc = myBlockNum;
  if (myBlockSelected>0) {
    Q = selectedBlock;
    blocks2calc = myBlockSelected;
    validOffset = totalNumBlock - numSelected;
  }
  const int dup = static_cast<int>(
    ceil(static_cast<real>(blocks2calc)/maxGrid));
  //if (dup>1) std::cerr << "DUP" << dup << std::endl;
  for (int _i=0;_i<dup;++_i) {
    _mpnum.x = (blocks2calc>(_i+1)*maxGrid) ? maxGrid : blocks2calc%maxGrid;
    //std::cerr << "myBlock: " << _i << " " << _mpnum.x << std::endl;
    calcF_IJpairWithBlock_F4<class SPHNavierStokes<SPHKernelLucyDW> ><<<_mpnum, _thnum>>>(P, r,
      tmp81N,
      validOffset + myOffsetSelected + maxGrid*_i,
      blockNeighbor, pid, bindex,
      N,
      NULL, // torque
      Q,    // selected block
      false,  // sorted r[] and typeid[]
      sortedOutput  // calcluated acceleration is sorted
    );
  }

  reduce27_F4<<<MPnum, THnum1D>>>(a, tmp81N, N);
#else
  dim3 _mpnum, _thnum;
  _thnum.x = THnum2D; _thnum.y = THnum2D; _thnum.z = 1;
  // thnum * thnum limitted by GPU's `threads per block'
  assert(_thnum.x*_thnum.y <= threadsMax);

  _mpnum.x = N / _thnum.x; _mpnum.y = N / _thnum.y ; _mpnum.z = 1;
  if ((_mpnum.x * _thnum.x) < N) ++_mpnum.x;
  if ((_mpnum.y * _thnum.y) < N) ++_mpnum.y;

//  std::cerr << "calculating cuda kernel with " << _mpnum.x << ":" << _mpnum.y << " blocks" << std::endl;
  calcF_SPH_NS<<<_mpnum, _thnum>>>(r, a, typeID, N, dW2D, rhoinv, m, mu0, v, 1.5e4);
#endif
  if (withInfo) ErrorInfo("calc acceleration for SPH_NS on GPU");
}

void cudaParticleSPH_NS::calcDensity(bool sortedOutput) {
  clearArray_F4<<<MPnum, THnum1D>>>(num, N);
  clearArray<<<MPnum, THnum1D>>>(rhoinv, N);
  if (withInfo) ErrorInfo("clear Array num, rho, rhoinv");

#if defined(CUDACUTOFFBLOCK)
  const uint32_t calcBlock = (sortedOutput) ? myBlockNum : totalNumBlock;
  dim3 _mpnum, _thnum;
  //_thnum.x = THnum2D; _thnum.y = THnum2D; _thnum.z = 1;
  _thnum.x = 8; _thnum.y = 9; _thnum.z = 1;
  assert(_thnum.x*_thnum.y <= threadsMax);
  _mpnum.x = calcBlock; _mpnum.y = 3; _mpnum.z = 1;
  clearArray<<<MPnum, THnum1D>>>(tmp81N, N*81);
  class SPHcalcDensity<SPHKernelLucy> P;
  P.m = m;
  P.h = h; P.w0 =w0;
  P.opt = mu0;

  const int dup = static_cast<int>(
    ceil(static_cast<real>(calcBlock)/maxGrid));
  for (int _i=0;_i<dup;++_i) {
    _mpnum.x = (calcBlock>(_i+1)*maxGrid) ? maxGrid : calcBlock%maxGrid;
    //std::cerr << "myBlock: " << _i << " " << _mpnum.x << std::endl;
    calcF_IJpairWithBlock_F4<SPHcalcDensity<SPHKernelLucy> ><<<_mpnum, _thnum>>>(P, r,
      tmp81N,
      ((sortedOutput) ? myBlockOffset : 0)+maxGrid*_i,
      blockNeighbor, pid, bindex,
      N,
      NULL, // torque
      NULL, // selected block
      false,  // sorted r[] and typeid[]
      sortedOutput  // calcluated acceleration is sorted
    );
  }
  if (withInfo) ErrorInfo("calc Density with Cutoff Block (SPH_NS)");

  reduce27_F4<<<MPnum, THnum1D>>>(num, tmp81N, N); //mu should be placed on num.z
  // re-pointing by num, mu is needed after the rho<->tmp3N swapping

#else
  std::cerr << "SPH_NS::calcDensity without cutoff Block is not implemented" << std::endl;
  exit(0);
#endif

  // no sorted output, no exchange
  if (!sortedOutput) calcDensityPost(false);
}

void cudaParticleSPH_NS::calcDensityPost(bool sortedOutput) {
  if (sortedOutput) {
    RestoreByPid_F4<<<MPnum, THnum1D>>>(tmp3N, num, N, pid);
    std::swap(num, tmp3N);
  }

  // rhoinv = 1.0/rho
  //calcReciproc_y<<<MPnum, THnum1D>>>(num, rhoinv, N);
  calcBinaryFunc<<<MPnum, THnum1D>>>(num, rhoinv, N, [] __device__ (float4 src, float &dst) {
    float4 tmp = src;
    assert((tmp.y!=0));
    dst = 1 / tmp.y;
  });
  if (withInfo) ErrorInfo("calc reciprocal of rho for SPH");

  // calculate field value mu from (\mu \rho)*\rho ^-1
  //multiplies_z<<<MPnum, THnum1D>>>(num, rhoinv, N); // mu is on num.z
  calcBinaryFunc<<<MPnum, THnum1D>>>(rhoinv, num, N, [] __device__ (float src, float4 &dst) {
    float4 tmp = dst;
    tmp.z *= src;
    dst = tmp;
  }); // mu is on num.z
  if (withInfo) ErrorInfo("\\mu\\rho *= \\rho^-1");
}

real cudaParticleSPH_NS::inspectDensity(void) {
  inspectDense_x<<<1, 128, 128*(sizeof(real)+sizeof(uint32_t))>>>(num, move, N, tmp3N);
  if (withInfo) ErrorInfo("inspect densities for SPH");
  real R[4]={1.0, 1.0, 1.0, 1.0};
  hipMemcpy(&R, tmp3N, sizeof(float4), hipMemcpyDeviceToHost);
  //std::cerr << "mean num density: " << R[0] << "," << R[1] << "," << R[2] << std::endl;
  return R[0];
}

void cudaParticleSPH_NS::RestoreAcceleration(void) {
  RestoreByPid_F4<<<MPnum, THnum1D>>>(tmp3N, a, N, pid);
  //hipDeviceSynchronize();
  std::swap(a, tmp3N);

  if (withInfo) ErrorInfo("RestoreByPid from a to tmp3N");
}

void cudaParticleSPH_NS::getExchangePidRange1(void) {
  uint32_t _p1 = myBlockOffset;
  uint32_t _p2 = _p1 + myBlockNum;

  bool _found=false;
  int trial=0;
  int __size=___p1.size();
  do {
    hipMemcpy(&___p1[0], &(bindex[_p1]),  sizeof(uint32_t)*__size, hipMemcpyDeviceToHost);
    std::vector<uint32_t>::iterator _ix = std::find_if(___p1.begin(), ___p1.begin()+__size, [](uint32_t i) {return i!=UINT_MAX;} );
    if (_ix!=___p1.end()) {
      p1 = *_ix;
      _found = true;
    } else {
      _p1 = myBlockOffset + ___p1.size() + trial * BINDEX_SEARCH_WIDTH;
      __size = BINDEX_SEARCH_WIDTH;
      ++trial;
    }
  } while(!_found);
  if (trial>0) ___p1.resize(___p1.size()+(trial)*BINDEX_SEARCH_WIDTH);

  if (_p2==totalNumBlock) p2 = N;
  else {
    _found=false;
    trial=0;
    __size=___p2.size();
    do {
      hipMemcpy(&___p2[0], &(bindex[_p2]),  sizeof(uint32_t)*__size, hipMemcpyDeviceToHost);
      std::vector<uint32_t>::iterator _ix = std::find_if(___p2.begin(), ___p2.begin()+__size, [] (uint32_t i) {return i!=UINT_MAX;} );
      if (_ix!=___p2.end()) {
        p2 = *_ix;
        _found = true;
      } else {
        _p2 = myBlockOffset + myBlockNum + ___p2.size() + trial * BINDEX_SEARCH_WIDTH;
        __size = BINDEX_SEARCH_WIDTH;
        ++trial;
      }
    } while(!_found);
    if (trial>0) ___p2.resize(___p2.size()+(trial)*BINDEX_SEARCH_WIDTH);
  }

/*
#pragma omp critical
  {
    std::cerr << myBlockOffset << ":" << myBlockOffset+myBlockNum << "[" << p1 << ":" << p2 << "]"
              << "(" << ___p1.size() << ":" << ___p2.size() << ")"
              << std::endl;
  }
*/

  if (withInfo) ErrorInfo("getExchangePidRange1");
}
void cudaParticleSPH_NS::getExchangePidRange2(void) {
  uint32_t _p3 = (totalNumBlock-numSelected)+myOffsetSelected;
  uint32_t _p4 = _p3 + myBlockSelected;
  const uint32_t _p5 = _p4;
  hipMemcpy(&_p3, &(selectedBlock[_p3]),  sizeof(uint32_t), hipMemcpyDeviceToHost);
  if (_p5<totalNumBlock)
    hipMemcpy(&_p4, &(selectedBlock[_p4]),  sizeof(uint32_t), hipMemcpyDeviceToHost);
  hipMemcpy(&_p3, &(bindex[_p3]),  sizeof(uint32_t), hipMemcpyDeviceToHost);
  if (_p5<totalNumBlock)
    hipMemcpy(&_p4, &(bindex[_p4]),  sizeof(uint32_t), hipMemcpyDeviceToHost);
  p3 = _p3;
  p4 = (_p5<totalNumBlock) ? _p4 : N;
  //std::cerr << p3 << ":" << p4 << std::endl;
  if (withInfo) ErrorInfo("getExchangePidRange2");
}

// override methods of cudaSelectedBlock
void cudaParticleSPH_NS::getForceSelected(const ExchangeMode typeID) {
  if (typeID!=ExchangeMode::density) cudaSelectedBlock::getForceSelected(typeID);
  else {
    size_t sizeN = sizeof(float4) * (p2-p1);

    pthread_mutex_lock(&mutTMP);
    hipMemcpy(&(TMP[p1]), num, sizeN, hipMemcpyDeviceToHost);
    pthread_mutex_unlock(&mutTMP);

    if (withInfo) ErrorInfo("cudaParticleSPH_NS::getForceSelected");
  }
}
void cudaParticleSPH_NS::importForceSelected(const cudaParticleSPH_NS &A, const ExchangeMode typeID,
    bool directAccess, int idMe, int idPeer) {
  if (typeID!=ExchangeMode::density) cudaSelectedBlock::importForceSelected(A, typeID, directAccess, idMe, idPeer);
  else {
    const size_t sizeN = sizeof(float4) * (A.p2-A.p1);
    if (directAccess) {
      hipMemcpyPeer(&(num[A.p1]),     idMe, &(A.num[A.p1]),     idPeer, sizeN);
      hipDeviceSynchronize();
    } else {
      hipMemcpy(&(num[A.p1]),     &(A.TMP[A.p1]),     sizeN, hipMemcpyHostToDevice);
    }

    if (withInfo) ErrorInfo("cudaParticleSPH_NS::importForceSelected");
  }
}
