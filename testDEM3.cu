#include "hip/hip_runtime.h"
#include <iostream>
#include "CUDAenv.hh"
#include "cudaParticleDEM.hh"
#include <fstream>
#include <boost/archive/binary_oarchive.hpp>
#include <math.h>
#include <random>

typedef std::vector<class ParticleBase> GlobalTable;

void createInitialState(cudaParticleDEM &particles) {
  /*
   * units of this simulation is:
   * length: cm
   * mass: g
   * time: s
   */

  std::mt19937 engine;
  
  const real R0=0.65;
  GlobalTable G1;
  const real L1 = 40.0;
  const real L2 = 100.0;
  const real L3 = L1 * 0.6;
  const real lunit = 2 * R0;
  const int lsize = L1 / (lunit) - 1;
  const int lsize2 = L2 / (lunit) - 1;
  const int lsize3 = L3 / (lunit) - 1;
  real cell[6] = {0.0, L2, 0.0, L1, 0.0, L3*1+9};

  real WeighFe = 7.874 * 4.0 / 3.0 * M_PI * R0*R0*R0;

/*
  for (int i=2;i<lsize-2;i+=1)
    for (int j=2;j<lsize-2;j+=1)
      for (int k=2+4;k<lsize-2+4;k+=1) {
*/
  std::normal_distribution<> s1(0.0, R0 / 10.0);
  const int P1 = (lsize - 4) / 1.2;
  const int P2 = L3 * 1 / lunit;
  for (int i=0;i<P1;i+=1)
    for (int j=0;j<P1;j+=1)
      for (int k=0;k<P2;k+=1) {
    ParticleBase pb;
	  pb.r[0] = (i*1.2  )*lunit+lunit/2 + s1(engine) + 35.0;
	  pb.r[1] = (j*1.2+2)*lunit+lunit/2 + s1(engine);
	  pb.r[2] = (k      )*lunit+lunit/2 + 9.0;
    pb.m = WeighFe;
    pb.v[0] = pb.v[1] = pb.v[2] = 0.0;
    pb.a[0] = 0.0; pb.a[1] = 0.0; pb.a[2] = 0.0;
    pb.isFixed = false;
    pb.type = 0;
    G1.push_back(pb);
      }
  uint32_t N1=G1.size();
  std::cerr << "N= " << N1 << std::endl;


  // border
  std::normal_distribution<> s2(0.0, R0 / 100.0);
  for (int i=0;i<lsize2;++i)
    for (int j=0;j<lsize;++j)
      for (int k=0;k<lsize3;++k) {
	if (( ((i<1)||(lsize2-2<i) ||
	       (j<1)||(lsize-2<j )) && (k>3*log(1.8-1.6*(real)(i)/(real)(lsize2))+5) )
	    ||
	    ((k==0) && ((0<i)&&(i<lsize2-1)&&(0<j)&&(j<lsize-1)) )
          ) {
	  ParticleBase pb;
	  pb.r[0] = i*lunit+lunit/2 + s2(engine);
	  pb.r[1] = j*lunit+lunit/2 + s2(engine);
	  if (k==0) 
	  pb.r[2] = (3*log(1.8-1.6*(real)(i)/(real)(lsize2))+5)*lunit+lunit/2;
	  else
	  pb.r[2] = k*lunit+lunit/2;
	  if (k<1) pb.r[2] += k*lunit+lunit/2;
	  pb.m = WeighFe;
	  pb.v[0] = pb.v[1] = pb.v[2] = 0.0;
	  pb.a[0] = 0.0; pb.a[1] = 0.0; pb.a[2] = 0.0;
	  pb.isFixed = true;
	  pb.type = 1;
	  G1.push_back(pb);
	}
      }
  uint32_t N = G1.size();
  std::cerr << "N= " << N << std::endl;



  particles.setup(N);
  particles.setCell(cell);

  particles.import(G1);
  //particles.setDEMProperties(1.0e11, 0.5, 0.3, 0.9, 0.2,
  //particles.setDEMProperties(1.0e5, 0.10, 0.45, 0.9, 0.05,
  // Fe: Young Modulus 211GPa, Poisson ratio 0.29, density 7.874gcm-3
  // 211GPa = 2.11x10^12 g cm^-1 sec^-2
  particles.setDEMProperties(2.11e07, 0.40, 0.29, 0.10, 0.10,
			     R0);
  particles.setInertia(R0);
  particles.setupCutoffBlock(R0*2/sqrt(3.0), false);

  // putTMPselected
  particles.setupSelectedTMP(0, N1, N1, N-N1);
  particles.putUnSelected("DEM3box.dump");

  particles.checkPidRange(0, 1, 0, N1);
}

int main(void) {


  class cudaParticleDEM particles;
  {
    CUDAenv<cudaParticleDEM> c;
    c.setThnum(particles);
  }

  createInitialState(particles);




  // calculate densities for the first output
  particles.calcBlockID();

  particles.getSelectedTypeID();
  particles.timestep = 0;
  particles.getSelectedPosition();
  particles.putTMP(std::cout);

  const real deltaT = 0.0000030;
  const uint32_t stepmax = 1.50 / deltaT;
  const uint32_t intaval  = 0.005 / deltaT;
  const uint32_t initstep = particles.timestep;

  particles.selectBlocks();
  particles.calcVinit(deltaT);

  for (uint32_t j=0;j<stepmax;++j) {
    if ((j%50)==0)
    std::cerr << j << " ";
    particles.calcBlockID();


    particles.selectBlocks();

    particles.calcForce(deltaT);
    particles.calcAcceleration();
    particles.addAccelerationZ(-9.8e2);
    particles.TimeEvolution(deltaT);
    particles.treatAbsoluteCondition();
    if ((j+1)%intaval==0) {
      particles.timestep = j+1+initstep;
      particles.getSelectedPosition();
      particles.putTMP(std::cout);
    }
  }

  std::ofstream ofs("DEM3done");
  boost::archive::binary_oarchive oa(ofs);
  oa << boost::serialization::make_nvp("cudaParticles", particles);
  ofs.close();

  return 0;
}
